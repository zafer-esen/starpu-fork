#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2009, 2010  Université de Bordeaux
 * Copyright (C) 2010, 2012  Centre National de la Recherche Scientifique
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>

static __global__ void cuda_incrementer(unsigned *token)
{
	(*token)++;
}

extern "C" void increment_cuda(void *descr[], void *_args)
{
	(void) _args;
	unsigned *tokenptr = (unsigned *)STARPU_VECTOR_GET_PTR(descr[0]);

	cuda_incrementer<<<1,1, 0, starpu_cuda_get_local_stream()>>>(tokenptr);
	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
