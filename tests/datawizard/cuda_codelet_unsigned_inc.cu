#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010  Université de Bordeaux
 * Copyright (C) 2010, 2012  Centre National de la Recherche Scientifique
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>

static __global__ void _cuda_unsigned_inc(unsigned *val)
{
	val[0]++;
}

extern "C" void cuda_codelet_unsigned_inc(void *descr[], STARPU_ATTRIBUTE_UNUSED void *cl_arg)
{
	unsigned *val = (unsigned *)STARPU_VARIABLE_GET_PTR(descr[0]);

	_cuda_unsigned_inc<<<1,1, 0, starpu_cuda_get_local_stream()>>>(val);

	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
