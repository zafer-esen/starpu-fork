#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010-2013  Centre National de la Recherche Scientifique
 * Copyright (C) 2010-2013  Université de Bordeaux
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

//! [To be included]
/* CUDA implementation of the `vector_scal' task, to be compiled with `nvcc'. */

#include <starpu.h>
#include <stdlib.h>

static __global__ void
vector_mult_cuda (unsigned n, float *val, float factor)
{
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n)
    val[i] *= factor;
}

/* Definition of the task implementation declared in the C file. */
extern "C" void
vector_scal_cuda (size_t size, float vector[], float factor)
{
  unsigned threads_per_block = 64;
  unsigned nblocks = (size + threads_per_block - 1) / threads_per_block;

  vector_mult_cuda <<< nblocks, threads_per_block, 0,
    starpu_cuda_get_local_stream () >>> (size, vector, factor);

  hipStreamSynchronize (starpu_cuda_get_local_stream ());
}
//! [To be included]
