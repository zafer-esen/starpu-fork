#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2012 Institut National de Recherche en Informatique et Automatique
 * Copyright (C) 2010, 2011, 2013  Centre National de la Recherche Scientifique
 * Copyright (C) 2010  Université de Bordeaux
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

/* CUDA implementation of the `vector_scal' task.  */

#include <starpu.h>
#include <starpu_cuda.h>
#include <stdlib.h>

static __global__ void
vector_mult_cuda (unsigned int n, float *val, float factor)
{
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n)
    val[i] *= factor;
}

extern "C" void
vector_scal_cuda (unsigned int size, float vector[], float factor)
{
  unsigned threads_per_block = 64;
  unsigned nblocks = (size + threads_per_block - 1) / threads_per_block;

  vector_mult_cuda <<< nblocks, threads_per_block, 0,
       starpu_cuda_get_local_stream () >>> (size, vector, factor);

  hipStreamSynchronize (starpu_cuda_get_local_stream ());
}
