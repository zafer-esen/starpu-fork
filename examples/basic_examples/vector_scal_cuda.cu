#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010, 2011, 2012, 2013  Centre National de la Recherche Scientifique
 * Copyright (C) 2010  Université de Bordeaux
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

/*
 * This example complements vector_scal.c: here we implement a CUDA version.
 */

#include <starpu.h>

static __global__ void vector_mult_cuda(unsigned n, float *val,
                                        float factor)
{
        unsigned i =  blockIdx.x*blockDim.x + threadIdx.x;

	if (i < n)
               val[i] *= factor;
}

extern "C" void scal_cuda_func(void *buffers[], void *_args)
{
        float *factor = (float *)_args;

        /* length of the vector */
        unsigned n = STARPU_VECTOR_GET_NX(buffers[0]);
        /* local copy of the vector pointer */
        float *val = (float *)STARPU_VECTOR_GET_PTR(buffers[0]);
	unsigned threads_per_block = 64;
	unsigned nblocks = (n + threads_per_block-1) / threads_per_block;

        vector_mult_cuda<<<nblocks,threads_per_block,0,starpu_cuda_get_local_stream()>>>(n, val, *factor);

	hipStreamSynchronize(starpu_cuda_get_local_stream());
}
