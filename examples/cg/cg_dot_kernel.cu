#include "hip/hip_runtime.h"
/* StarPU --- Runtime system for heterogeneous multicore architectures.
 *
 * Copyright (C) 2010, 2015  Université de Bordeaux
 * Copyright (C) 2010, 2012  Centre National de la Recherche Scientifique
 *
 * StarPU is free software; you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation; either version 2.1 of the License, or (at
 * your option) any later version.
 *
 * StarPU is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Lesser General Public License in COPYING.LGPL for more details.
 */

#include <starpu.h>

#include "cg.h"

#define MAXNBLOCKS	128
#define MAXTHREADSPERBLOCK	256

/*
 * Dot product kernel
 * We first perform dot computation in parallel in dot_device, and then we
 * gather the dot values into one in gather_dot_device.
 */

static __global__ void dot_device(TYPE *vx, TYPE *vy, unsigned n, TYPE *dot_array)
{
	__shared__ TYPE scnt[MAXTHREADSPERBLOCK];

	/* Do we have a successful shot ? */
	const int tid = threadIdx.x + blockIdx.x*blockDim.x;

	const int nthreads = gridDim.x * blockDim.x;

	/* Blank the shared mem buffer */
	if (threadIdx.x < MAXTHREADSPERBLOCK)
		scnt[threadIdx.x] = (TYPE)0.0;

	__syncthreads();

	int ind;
	for (ind = tid; ind < n; ind += nthreads)
	{ 
		TYPE x = vx[ind];
		TYPE y = vy[ind];

		scnt[threadIdx.x] += (x*y);
	}

	__syncthreads();

	/* Perform a reduction to compute the sum on each thread within that block */

	/* NB: We assume that the number of threads per block is a power of 2 ! */
	unsigned s;
	for (s = blockDim.x/2; s!=0; s>>=1)
	{
		if (threadIdx.x < s)
			scnt[threadIdx.x] += scnt[threadIdx.x + s];

		__syncthreads();
	}

	/* report the number of successful shots in the block */
	if (threadIdx.x == 0)
		dot_array[blockIdx.x] = scnt[0];

	__syncthreads();
}

static __global__ void gather_dot_device(TYPE *dot_array, TYPE *dot)
{
	__shared__ TYPE accumulator[MAXNBLOCKS];

	unsigned i;

	/* Load the values from global mem */
	for (i = 0; i < blockDim.x; i++)
		accumulator[i] = dot_array[i];

	__syncthreads();

	/* Perform a reduction in shared memory */
	unsigned s;
	for (s = blockDim.x/2; s!=0; s>>=1)
	{
		if (threadIdx.x < s)
			accumulator[threadIdx.x] += accumulator[threadIdx.x + s];

		__syncthreads();
	}


	/* Save the result in global memory */
	if (threadIdx.x == 0)
		*dot = *dot + accumulator[0];
}

extern "C" void dot_host(TYPE *x, TYPE *y, unsigned nelems, TYPE *dot)
{
	/* How many blocks do we use ? */ 
	unsigned nblocks = 128; // TODO
	STARPU_ASSERT(nblocks <= MAXNBLOCKS);
	
	TYPE *per_block_sum;
	hipMalloc((void **)&per_block_sum, nblocks*sizeof(TYPE));

	STARPU_ASSERT((nelems % nblocks) == 0);

	/* How many threads per block ? At most 256, but no more threads than
	 * there are entries to process per block. */
	unsigned nthread_per_block = STARPU_MIN(MAXTHREADSPERBLOCK, (nelems / nblocks));

	/* each entry of per_block_sum contains the number of successful shots
	 * in the corresponding block. */
	dot_device<<<nblocks, nthread_per_block, 0, starpu_cuda_get_local_stream()>>>(x, y, nelems, per_block_sum);

	/* Note that we do not synchronize between kernel calls because there
	 * is an implicit serialization */
	gather_dot_device<<<1, nblocks, 0, starpu_cuda_get_local_stream()>>>(per_block_sum, dot);

	hipError_t cures;
	cures = hipStreamSynchronize(starpu_cuda_get_local_stream());
	if (cures)
		STARPU_CUDA_REPORT_ERROR(cures);

	hipFree(per_block_sum);
}

/*
 * Fill a vector with zeroes
 */

static __global__ void zero_vector_device(TYPE *x, unsigned nelems, unsigned nelems_per_thread)
{
	unsigned i;
	unsigned first_i = blockDim.x * blockIdx.x + threadIdx.x;

	for (i = first_i; i < nelems; i += nelems_per_thread)
		x[i] = 0.0;
}

extern "C" void zero_vector(TYPE *x, unsigned nelems)
{
	unsigned nblocks = STARPU_MIN(128, nelems);
	unsigned nthread_per_block = STARPU_MIN(MAXTHREADSPERBLOCK, (nelems / nblocks));

	unsigned nelems_per_thread = nelems / (nblocks * nthread_per_block);

	zero_vector_device<<<nblocks, nthread_per_block, 0, starpu_cuda_get_local_stream()>>>(x, nelems, nelems_per_thread);
}
